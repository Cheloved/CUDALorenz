#include "hip/hip_runtime.h"
#include "includes/defines.cuh"
#include "includes/lorenz.cuh"
#include <stdio.h>
#include <>


__global__ void update_points(vector* c_points)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if ( i >= POINTS_COUNT )
        return;

    vector p; vector_copy(&p, &c_points[i]);
    vector velocity = { .x = 10.0 * (p.y - p.x),
                        .y = p.x * (28.0 - p.z) - p.y,
                        .z = p.x * p.y - (8.0 / 3.0) * p.z};

    vector_normalize(&velocity, &velocity);
    vector_mul(&velocity, TIME_SCALE, &velocity);

    vector_add(&c_points[i], &velocity, &c_points[i]);

    return;
}

__global__ void points_to_pixel_space(vector* c_points,
                                      vector* c_normal, vector* c_orig, vector* c_center,
                                      vector* c_basis, s_settings* c_settings, vector* c_pixels)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if ( i >= POINTS_COUNT )
        return;

    // Get projection of point onto the plane
    //printf("c_points[i]: (%lf, %lf, %lf)\n", c_points[i].x, c_points[i].y, c_points[i].z);
    //printf("c_normal: (%lf, %lf, %lf)\n", c_normal->x, c_normal->y, c_normal->z);
    //printf("c_orig: (%lf, %lf, %lf)\n", c_orig->x, c_orig->y, c_orig->z);
    vector coords; get_projection(&c_points[i], c_normal, c_orig, &coords);
    //printf("After get_projection()\n");
    //printf("coords: (%lf, %lf, %lf)\n", coords.x, coords.y, coords.z);

    if ( c_settings->angleChanged )
    {
        vector_spherical_to_cartesian(c_settings->theta, c_settings->phi, 1.0, c_normal);
        //printf("theta, phi: (%lf, %lf)\n", c_settings->theta, c_settings->phi);
        vector_normalize(c_normal, c_normal);
        //printf("c_normal: (%lf, %lf, %lf)\n", c_normal->x, c_normal->y, c_normal->z);

		// Find the center of the plane
        vector_sub(c_center, c_normal, c_orig);
        //printf("c_orig: (%lf, %lf, %lf)\n", c_orig->x, c_orig->y, c_orig->z);

		// Create a basis of the plane
        get_basis(c_normal, c_orig, c_basis);
        //printf("Basis:\ne1 (%lf, %lf, %lf)\ne2 (%lf, %lf, %lf)\ne3 (%lf, %lf, %lf)\n",
        //        c_basis[0].x, c_basis[0].y, c_basis[0].z,
        //        c_basis[1].x, c_basis[1].y, c_basis[1].z,
        //        c_basis[2].x, c_basis[2].y, c_basis[2].z);

        c_settings->angleChanged = 0;
    }
    //printf("After condition\n");
    //printf("Basis:\ne1 (%lf, %lf, %lf)\ne2 (%lf, %lf, %lf)\ne3 (%lf, %lf, %lf)\n",
    //        c_basis[0].x, c_basis[0].y, c_basis[0].z,
    //        c_basis[1].x, c_basis[1].y, c_basis[1].z,
    //        c_basis[2].x, c_basis[2].y, c_basis[2].z);
    //printf("coords: (%lf, %lf, %lf)\n", coords.x, coords.y, coords.z);

    // Decompose coordinates by plane basis
    // so that z coordinate is equal to zero
    decompose_by_basis(&coords, c_basis, &c_pixels[i]);
    //printf("Result: (%lf, %lf, %lf)\n", c_pixels[i].x, c_pixels[i].y, c_pixels[i].z);
    return;
}

__global__ void update_pixels(vector* c_pixels, s_settings* c_settings, float* c_pixel_buffer)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if ( i >= POINTS_COUNT )
        return;

    int x = (c_pixels[i].x + c_settings->offsetX) / c_settings->scale * SCREEN_WIDTH;
    int y = (c_pixels[i].y + c_settings->offsetY) / c_settings->scale * SCREEN_HEIGHT;

    int base_idx = (y*SCREEN_WIDTH + x) * 3;

    if ( base_idx < 0 || base_idx > SCREEN_HEIGHT*SCREEN_WIDTH * 3 )
        return;

    c_pixel_buffer[ base_idx + 0] = 1;
    c_pixel_buffer[ base_idx + 1] = 1;
    c_pixel_buffer[ base_idx + 2] = 1;

    return;
}

__global__ void clear_buffer(float* c_pixel_buffer)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if ( i >= SCREEN_WIDTH * SCREEN_HEIGHT )
        return;

    c_pixel_buffer[i*3 + 0] = 0;
    c_pixel_buffer[i*3 + 1] = 0;
    c_pixel_buffer[i*3 + 2] = 0;

    return;
}
